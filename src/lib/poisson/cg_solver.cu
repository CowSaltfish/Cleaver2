#include "hip/hip_runtime.h"
//=========================================================================
//  cg_solver.cu
//
//  This file contains the cuda code for the cg solver
//
//  Author: Jihwan Kim
//  CS6963, Spring 2011
//  Final Project
//=========================================================================

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "cg_constants.h"


__device__ void laplacian2DSharedTwo(int index, int tileNum, float (&sx)[SHARED_WIDTH][SHARED_WIDTH], 
              float (&sx_next)[TILE_WIDTH][TILE_WIDTH], float* x, int oneSideLength, int by, int tx, int ty){
   float val = 0;
              
      if(tileNum == 0 ){
         sx[ty+1][tx+1] = x[index];

         syncthreads();
                  
         if(ty == 0){
            if(by == 0){
               sx[0][tx+1] = 0;//sx[2][tx+1]; //top
            }else{
               sx[0][tx+1] = x[index - oneSideLength];
            }
         }
         
         if(ty == TILE_WIDTH -1){            
            sx[tx+1][0] = 0;//sx[tx+1][2]; //left
            if(by < gridDim.y-1){
               sx[TILE_WIDTH + 1][tx+1] = x[index + oneSideLength];//bottom
            }else{
               sx[TILE_WIDTH + 1][tx+1] = 0;//sx[TILE_WIDTH-1][tx+1];
            }
         }
      }else{
         if(ty == 0){
            sx[tx+1][0] = sx[tx+1][TILE_WIDTH]; //left
         }
         
         syncthreads();
         sx[ty+1][tx+1] = sx_next[ty][tx];

         syncthreads();
         if(ty == 0){
            if(by == 0){
               sx[0][tx+1] = 0;//sx[2][tx+1]; //top
            }else{
               sx[0][tx+1] = x[index - oneSideLength];
            }            
         }
         
         if(ty == TILE_WIDTH -1){
            if(by < gridDim.y-1){
               sx[TILE_WIDTH + 1][tx+1] = x[index + oneSideLength];//bottom
            }else{
               sx[TILE_WIDTH + 1][tx+1] = 0;//sx[TILE_WIDTH-1][tx+1];
            }
         }
         
      }
      
      if(tileNum < oneSideLength/TILE_WIDTH - 1){
         val = x[index+TILE_WIDTH];
         sx_next[ty][tx] = val;
         if(tx == 0){
            sx[ty+1][TILE_WIDTH +1] = val;//sx_next[ty][0];
         }
         
      }else if(ty == (TILE_WIDTH -1)){
         sx[tx+1][TILE_WIDTH+1] = 0;//sx[tx+1][TILE_WIDTH-1];//right
      }
      /*
      syncthreads();

      if(ty == (TILE_WIDTH -1) && tileNum < oneSideLength/TILE_WIDTH - 1){
         sx[tx+1][TILE_WIDTH+1] = sx_next[tx][0]; //right
      }else if(ty == (TILE_WIDTH -1) && tileNum == oneSideLength/TILE_WIDTH - 1){
         sx[tx+1][TILE_WIDTH+1] = 0;//sx[tx+1][TILE_WIDTH-1];//right
      }
      */
      syncthreads();
      
}



__device__ void laplacian2DShared(int index, int tileNum, float (&sx)[SHARED_WIDTH][SHARED_WIDTH], 
              /*float (&sx_next)[TILE_WIDTH][TILE_WIDTH],*/ float* x, int oneSideLength, int by, int tx, int ty){
              
      if(tileNum == 0 ){
         sx[ty+1][tx+1] = x[index];

         syncthreads();
                  
         if(ty == 0){
            if(by == 0){
               sx[0][tx+1] = 0;//sx[2][tx+1]; //top
            }else{
               sx[0][tx+1] = x[index - oneSideLength];
            }
         }
         
         if(ty == TILE_WIDTH -1){            
            sx[tx+1][0] = 0;//sx[tx+1][2]; //left
            if(by < gridDim.y-1){
               sx[TILE_WIDTH + 1][tx+1] = x[index + oneSideLength];//bottom
            }else{
               sx[TILE_WIDTH + 1][tx+1] = 0;//sx[TILE_WIDTH-1][tx+1];
            }
         }
      }else{
         if(ty == 0){
            sx[tx+1][0] = sx[tx+1][TILE_WIDTH]; //left
         }
         
         syncthreads();
         sx[ty+1][tx+1] = x[index];   //sx_next[ty][tx];

         syncthreads();
         if(ty == 0){
            if(by == 0){
               sx[0][tx+1] = 0;//sx[2][tx+1]; //top
            }else{
               sx[0][tx+1] = x[index - oneSideLength];
            }            
         }
         
         if(ty == TILE_WIDTH -1){
            if(by < gridDim.y-1){
               sx[TILE_WIDTH + 1][tx+1] = x[index + oneSideLength];//bottom
            }else{
               sx[TILE_WIDTH + 1][tx+1] = 0;//sx[TILE_WIDTH-1][tx+1];
            }
         }
         
      }
      /*
      if(tileNum < oneSideLength/TILE_WIDTH - 1){
         sx_next[ty][tx] = x[index+TILE_WIDTH];
      }*/
             
//      syncthreads();

      if(ty == (TILE_WIDTH -1) && tileNum < oneSideLength/TILE_WIDTH - 1){
         int ind=index-tx + (TILE_WIDTH);   
         
         sx[tx+1][TILE_WIDTH+1] = x[ind - (TILE_WIDTH-tx-1)*oneSideLength];    //sx_next[tx][0]; //right
      }else if(ty == (TILE_WIDTH -1) && tileNum == oneSideLength/TILE_WIDTH - 1){
         sx[tx+1][TILE_WIDTH+1] = 0;//sx[tx+1][TILE_WIDTH-1];//right
      }
            
      syncthreads();
      
}


__global__ void residual_init_laplacian2D(float *b, float* x, float* r, float* innerR, int oneSideLength, int tileWith_oneSideWidth){
   __shared__ float sx[SHARED_WIDTH][SHARED_WIDTH];
//   __shared__ float sx_next[TILE_WIDTH][TILE_WIDTH];
   
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int index = 0;
   float residual = 0;
   
   int ind = by*tileWith_oneSideWidth + ty*oneSideLength + tx;   
   
   for(int tileNum = 0; tileNum < oneSideLength/TILE_WIDTH; tileNum++){
      index = ind + tileNum*TILE_WIDTH; 
      
      laplacian2DShared(index, tileNum, sx, /*sx_next,*/ x, oneSideLength, by, tx, ty);
//      laplacian2DSharedTwo(index, tileNum, sx, sx_next, x, oneSideLength, by, tx, ty);      

      residual = b[index] + (sx[ty+1][tx] + sx[ty+1][tx+2] -4*sx[ty+1][tx+1] + sx[ty][tx+1] + sx[ty+2][tx+1]);

      r[index] = residual;
      innerR[index] = residual*residual;
   }
}



__global__ void laplacian2D(float* x, float* answer, float* answerSq, int oneSideLength, int tileWith_oneSideWidth){
   __shared__ float sx[SHARED_WIDTH][SHARED_WIDTH];
//   __shared__ float sx_next[TILE_WIDTH][TILE_WIDTH];
   
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int index = 0;
   float residual = 0;
   int ind = by*tileWith_oneSideWidth + ty*oneSideLength + tx;   
   for(int tileNum = 0; tileNum < oneSideLength/TILE_WIDTH; tileNum++){
      index = ind + tileNum*TILE_WIDTH;
      
      laplacian2DShared(index, tileNum, sx, /*sx_next,*/ x, oneSideLength, by, tx, ty);
//      laplacian2DSharedTwo(index, tileNum, sx, sx_next, x, oneSideLength, by, tx, ty);

      residual = -(sx[ty+1][tx] + sx[ty+1][tx+2] -4*sx[ty+1][tx+1] + sx[ty][tx+1] + sx[ty+2][tx+1]);

      answer[index] = residual;
      answerSq[index] = residual*sx[ty+1][tx+1];
   }
}


__global__ void arraySqSum(float* array, float* sumArray, int oneSideLength, int tileWith_oneSideWidth){
   __shared__ float sArray[TILE_WIDTH][TILE_WIDTH];
   __shared__ float sRowSum[TILE_WIDTH];

   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int index = 0;
   float rowSum = 0;

   int ind = by*tileWith_oneSideWidth + ty*oneSideLength + tx;   
   for(int tileNum = 0; tileNum < oneSideLength/TILE_WIDTH; tileNum++){
      index = ind + tileNum * TILE_WIDTH;
      sArray[ty][tx] = array[index] * array[index];


      for(unsigned int stride = blockDim.x >>1; stride >=1; stride = stride>>1){
         syncthreads();
         if(tx < stride){
            sArray[ty][tx] += sArray[ty][tx+stride];
         }
      }


      if(tx == 0){
         rowSum += sArray[ty][0];
      }

      syncthreads();
   }

   if(tx == 0){
      sRowSum[ty] = rowSum;
   }

   syncthreads();

   if(ty == 0){
      for(unsigned int stride = blockDim.x >>1; stride >=1; stride = stride>>1){
         syncthreads();
         if(tx < stride){
            sRowSum[tx] += sRowSum[tx+stride];
         }
      }

      syncthreads();

      if(tx == 0){
         sumArray[by] = sRowSum[0];
      }
   }
}



__global__ void arraySum(float* array, float* sumArray, int oneSideLength, int tileWith_oneSideWidth){
   __shared__ float sArray[TILE_WIDTH][TILE_WIDTH];
   __shared__ float sRowSum[TILE_WIDTH];
   
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int index = 0;
   float rowSum = 0;

   int ind = by*tileWith_oneSideWidth + ty*oneSideLength + tx;   
   for(int tileNum = 0; tileNum < oneSideLength/TILE_WIDTH; tileNum++){
      index = ind + tileNum * TILE_WIDTH;   
      sArray[ty][tx] = array[index];
      
      
      for(unsigned int stride = blockDim.x >>1; stride >=1; stride = stride>>1){
         syncthreads();
         if(tx < stride){
            sArray[ty][tx] += sArray[ty][tx+stride];
         }
      }


      if(tx == 0){
         rowSum += sArray[ty][0];
      }
         
      syncthreads();      
   }
   
   if(tx == 0){
      sRowSum[ty] = rowSum;
   }
   
   syncthreads();      
   
   if(ty == 0){
      for(unsigned int stride = blockDim.x >>1; stride >=1; stride = stride>>1){
         syncthreads();
         if(tx < stride){
            sRowSum[tx] += sRowSum[tx+stride];
         }
      }
      
      syncthreads();
      
      if(tx == 0){
         sumArray[by] = sRowSum[0];
      }
   }
}

//Should run by one block
__device__ float sumOfArray(int tx, int blockDimX, float* a){
   __shared__ float sInner[MAX_N/TILE_WIDTH];

   float innerSum = 0.0f;

   sInner[tx] = a[tx];

   syncthreads();
         
   for(unsigned int stride = blockDimX>>1; stride >=1; stride = stride>>1){
      syncthreads();
      if(tx < stride){
         sInner[tx] += sInner[tx+stride];
      }
   }

   if(tx == 0){
      innerSum += sInner[0];
   }
   
   return innerSum;
}


//Should run by one block
__global__ void arrayOneDSum(float* a, float* sum){
   float prod = 0;
   int tx = threadIdx.x;

   prod = sumOfArray(tx, blockDim.x, a);
   
   syncthreads();  
   
   if(tx == 0){
      sum[0]  = prod;
   }
}


__global__ void arrayOneDSum(float* a, float* nominator, float* sum){
   float prod = 0;
   int tx = threadIdx.x;

   prod = sumOfArray(tx, blockDim.x, a);
   
   syncthreads();  
   
   if(tx == 0){
      sum[0]  = nominator[0]/prod;
   }
}


__global__ void arrayOneDSum(float* a, float* denominator, float* sum, float* bnorm, float tolerance){
   float prod = 0;

   int tx = threadIdx.x;

   prod = sumOfArray(tx, blockDim.x, a);         
   syncthreads();  
   
   if(tx == 0){
      sum[0]  = prod/denominator[0];
      denominator[0] = prod;
      
      if(bnorm != NULL && prod/bnorm[0] < tolerance){
         bnorm[0] = -1;
      }
   }
}


__global__ void innerCG_XR(float* dr,  float* dx, float* dd, float* dq, float* drSq, float* dalpha, int oneSideLength, int tileWith_oneSideWidth){ 
   float alpha = dalpha[0];
   
   int index = blockIdx.y*tileWith_oneSideWidth + threadIdx.y*oneSideLength + blockIdx.x*TILE_WIDTH + threadIdx.x;

   dx[index] = dx[index] + alpha*dd[index];
   float newR = dr[index] - alpha*dq[index];
   dr[index] = newR;
   drSq[index] = newR*newR;
}


__global__ void innerCG_D(float* dr,  float* dd, float* dbeta, int oneSideLength, int tileWith_oneSideWidth){ 
   float beta = dbeta[0];
   
   int index = blockIdx.y*tileWith_oneSideWidth + threadIdx.y*oneSideLength + blockIdx.x*TILE_WIDTH + threadIdx.x;

   dd[index] = dr[index] + beta*dd[index];
}





float cg_solver(float* b, float* x, int max_iter, float tolerance, float* answer, const int width){

   float* db;
   float* dx;
   float* dresidual;
   float* danswerSq;
   float* sumPerBlock;
   float* dd;
   float* dq;
   float* ddelta;
   float* dbnorm;
   float* dalpha;
   float* dbeta;
   const int tileWith_oneSideWidth = width * TILE_WIDTH;
   
   hipEvent_t start_event, stop_event;
   float my_elapsed_time = 0;
   float sumTime = 0;
   
   hipError_t error;

   hipMalloc((void**)&db, sizeof(float) * width*width);
   hipMalloc((void**)&dx, sizeof(float) * width*width);
   hipMalloc((void**)&dresidual, sizeof(float) * width*width);
   hipMalloc((void**)&danswerSq, sizeof(float) * width*width);
   hipMalloc((void**)&dd, sizeof(float) * width*width);
   hipMalloc((void**)&dq, sizeof(float) * width*width);
   hipMalloc((void**)&sumPerBlock, sizeof(float)*width/TILE_WIDTH);
   
   hipMalloc((void**)&ddelta, sizeof(float));
   hipMalloc((void**)&dbnorm, sizeof(float));
   hipMalloc((void**)&dalpha, sizeof(float));
   hipMalloc((void**)&dbeta, sizeof(float));

   hipMemcpy(db, b, sizeof(float) * width*width, hipMemcpyHostToDevice);
   hipMemcpy(dx, x, sizeof(float) * width*width, hipMemcpyHostToDevice);



   //Performance Measure event
   CUDA_SAFE_CALL( hipEventCreate(&start_event));
   CUDA_SAFE_CALL( hipEventCreate(&stop_event));

   // start the timer for GPU code
   hipEventRecord(start_event, 0);


   dim3 dimGrid(1, width/TILE_WIDTH, 1);
   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

   residual_init_laplacian2D<<<dimGrid, dimBlock>>>(db, dx, dresidual, danswerSq, width, tileWith_oneSideWidth);
   error = hipDeviceSynchronize();

   error = hipGetLastError();
    if( hipSuccess != error)
    {
        printf("Cuda error: %s.\n", hipGetErrorString( error) );
        return 0;
    }

   arraySqSum<<<dimGrid, dimBlock>>>(db, sumPerBlock, width, tileWith_oneSideWidth);
   error = hipDeviceSynchronize(); 

   dimGrid.y = 1;
   dimBlock.x = (width/TILE_WIDTH);  //Assume width/TILE_WIDTH is less than 512 and width/TILE_WIDTH is also 2^n.
   dimBlock.y = 1;
   
   arrayOneDSum<<<dimGrid, dimBlock>>>(sumPerBlock, dbnorm);
   error = hipDeviceSynchronize();

   hipMemcpy(dd, dresidual, sizeof(float) * width*width, hipMemcpyDeviceToDevice);

   dimGrid.y = width/TILE_WIDTH;
   dimBlock.x = TILE_WIDTH;  
   dimBlock.y = TILE_WIDTH;
      
   arraySum<<<dimGrid, dimBlock>>>(danswerSq, sumPerBlock, width, tileWith_oneSideWidth);
   error = hipDeviceSynchronize();
   
   dimGrid.y = 1;
   dimBlock.x = (width/TILE_WIDTH);  //Assume width/TILE_WIDTH is less than 512 and width/TILE_WIDTH is also 2^n.
   dimBlock.y = 1;
   
   arrayOneDSum<<<dimGrid, dimBlock>>>(sumPerBlock, ddelta, dbeta, NULL, 0);
   error = hipDeviceSynchronize();

   int iter = 0;
   for (iter = 0; iter<max_iter; iter++){
           
      dimGrid.x = 1;
      dimGrid.y = width/TILE_WIDTH;

      dimBlock.x = TILE_WIDTH;  
      dimBlock.y = TILE_WIDTH;
      
      laplacian2D<<<dimGrid, dimBlock>>>(dd, dq, danswerSq, width, tileWith_oneSideWidth);
      error = hipDeviceSynchronize();

      arraySum<<<dimGrid, dimBlock>>>(danswerSq, sumPerBlock, width, tileWith_oneSideWidth);
      error = hipDeviceSynchronize();
   
      dimGrid.y = 1;
      dimBlock.x = (width/TILE_WIDTH);  //Assume width/TILE_WIDTH is less than 512 and width/TILE_WIDTH is also 2^n.
      dimBlock.y = 1;
   
      arrayOneDSum<<<dimGrid, dimBlock>>>(sumPerBlock, ddelta, dalpha);
      error = hipDeviceSynchronize();

      dimGrid.x = width/TILE_WIDTH;
      dimGrid.y = width/TILE_WIDTH;
      dimBlock.x = TILE_WIDTH;  
      dimBlock.y = TILE_WIDTH;
          
      innerCG_XR<<<dimGrid, dimBlock>>>(dresidual, dx, dd, dq, danswerSq, dalpha, width, tileWith_oneSideWidth);
      error = hipDeviceSynchronize();

      dimGrid.x = 1;
      arraySum<<<dimGrid, dimBlock>>>(danswerSq, sumPerBlock, width, tileWith_oneSideWidth);
      error = hipDeviceSynchronize();
   
      dimGrid.y = 1;
      dimBlock.x = (width/TILE_WIDTH);  //Assume width/TILE_WIDTH is less than 512 and width/TILE_WIDTH is also 2^n.
      dimBlock.y = 1;
   
      arrayOneDSum<<<dimGrid, dimBlock>>>(sumPerBlock, ddelta, dbeta, dbnorm, tolerance);
      error = hipDeviceSynchronize();
           
      hipMemcpy(answer, dbnorm, sizeof(float), hipMemcpyDeviceToHost);
      if(answer[0] < 0){
         break;
      }
      
      dimGrid.x = width/TILE_WIDTH;
      dimGrid.y = width/TILE_WIDTH;
      dimBlock.x = TILE_WIDTH;  
      dimBlock.y = TILE_WIDTH;
            
      innerCG_D<<<dimGrid, dimBlock>>>(dresidual, dd, dbeta, width, tileWith_oneSideWidth);
      error = hipDeviceSynchronize();       
   }

    // stop timing
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    CUDA_SAFE_CALL( hipEventElapsedTime(&my_elapsed_time, start_event, stop_event));

    std::cout<< "Break the loop after "<< iter <<" iteration" << std::endl;

    my_elapsed_time = my_elapsed_time/iter; //sumTime;
   
   hipMemcpy(answer, dx, sizeof(float) * width*width, hipMemcpyDeviceToHost);

/*
   printf("answer in the cuda: \n");
   for(i=0; i<width*width; i++){
       printf("%f, ", answer[i]);
   }
   printf("\n");
*/
   hipFree(db);
   hipFree(dx);
   hipFree(dresidual);
   hipFree(danswerSq);
   hipFree(sumPerBlock);
   hipFree(dd);
   hipFree(dq);
   hipFree(ddelta);
   hipFree(dbnorm);
   hipFree(dalpha);
   hipFree(dbeta);   

   return my_elapsed_time;
}

